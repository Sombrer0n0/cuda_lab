/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号:SA24221035
 * 姓名:商坤杰
 * 邮箱:sun_kj@mail.ustc.edu.cn
 ------------------------------------------------*/


#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>

#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]

using std::cin, std::cout, std::endl;
using std::ifstream, std::ofstream;

// 存活细胞数
int population(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

// 打印世界状态
void print_universe(int N, char *universe)
{
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, universe) << endl;
}

// kernel代码,计算下一状态
__global__ void life3d_kernel(unsigned char* current, unsigned char* next, int N)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	
	if (x>=N || y>N || z>=N) return; //N与THREADS_PER_DIM非整数倍情况会产生冗余线程的情况
	
	int idx = x*N*N + y*N + z;
	
	int alive = 0;
	for (int dx = -1; dx <= 1; dx++)
		for (int dy = -1; dy <= 1; dy++)
			for (int dz = -1; dz <= 1; dz++)
			{
				if (dx == 0 && dy == 0 && dz == 0)
					continue;
				int nx = (x + dx + N) % N;
				int ny = (y + dy + N) % N;
				int nz = (z + dz + N) % N;
				int n_idx = nx*N*N + ny*N + nz;
				alive += current[n_idx];
			}
	if (current[idx] && (alive < 5 || alive > 7))
		next[idx] = 0;
	else if (!current[idx] && alive == 6)
		next[idx] = 1;
	else
		next[idx] = current[idx];
}
// CUDA实现将世界推进T个时刻
void life3d_run_cuda(int N, char *universe, int T){
	size_t size = N*N*N*sizeof(unsigned char);
	
	unsigned char *d_current,*d_next;
	
	hipMalloc((void**)&d_current,size);
	hipMalloc((void**)&d_next,size);
	hipMemcpy(d_current,universe,size,hipMemcpyHostToDevice);
	
	int THREAD_PER_DIM = 8;
	dim3 threads(THREAD_PER_DIM,THREAD_PER_DIM,THREAD_PER_DIM);
	dim3 blocks((N-1)/threads.x + 1,(N-1)/threads.y + 1,(N-1)/threads.z + 1);
	
	for(int t = 0;t < T; t++){
		life3d_kernel<<<blocks,threads>>>(d_current,d_next,N); //计算每轮结果
		std::swap(d_current,d_next);
	}
	
	hipDeviceSynchronize;	//同步
	
	hipMemcpy(universe, d_current, size, hipMemcpyDeviceToHost);
	
	hipFree(d_current);
    hipFree(d_next);
}

// 读取输入文件
void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open())
    {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size))
    {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file)
    {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char **argv)
{
    // cmd args
    if (argc < 5)
    {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char *input_file = argv[3];
    char *output_file = argv[4];

    char *universe = (char *)malloc(N * N * N);
    read_file(input_file, universe);

    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    life3d_run_cuda(N, universe, T);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);
    write_file(output_file, universe, N);

    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;

    free(universe);
    return 0;
}
